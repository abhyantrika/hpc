
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <stdio.h>

__global__
void add(double *a,double*b,double* c,int n) {

    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id>0)
        c[id] = a[id] + b[id];

}

int main()
{

   int n = 100;

 // Device input vectors
    double *d_a;
    double *d_b;
    //Device output vector
    double *d_c;
 
    int i=0;

    hipMallocManaged(&d_a,n*sizeof(double));
    hipMallocManaged(&d_b,n*sizeof(double));
    hipMallocManaged(&d_c,n*sizeof(double));

     for ( i = 0; i < n; i++) {
        d_a[i] = i;
        d_b[i] = i;
      }


    int blockSize = 512;
    // Number of thread blocks in grid
    int gridSize = (int)ceil((float)n/blockSize);

    add <<< gridSize,blockSize >>>(d_a,d_b,d_c,n);
        hipDeviceSynchronize();

     printf("%d  %d\n",gridSize,blockSize );   

     for(i=0;i<n;i++)
     {
        printf("%f + %f = %f\n",d_a[i],d_b[i],d_c[i]);
     }  

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
     


    /*float maxError = 0.0f;
    for (int i = 0; i < n; i++)
    maxError = fmax(maxError, fabs(d_c[i]-3.0f));
    std::cout << "Max error: " << maxError << std::endl;*/

}
