
#include <hip/hip_runtime.h>
#include <stdio.h>
#define NUM_BLOCKS 3	//Number of blocks
#define BLOCK_WIDTH 192	//number of threads in in the thread block

__global__ void hello()
{
    printf("Hello world! I'm a thread in block %d and my thread id is %d\n", blockIdx.x,threadIdx.x);
}


int main(int argc,char **argv)
{
    // launch the kernel
    hello<<<NUM_BLOCKS, BLOCK_WIDTH>>>();

    // force the printf()s to flush
    hipDeviceSynchronize();

    printf("That's all!\n");

    return 0;
}

