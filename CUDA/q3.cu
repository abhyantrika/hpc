#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 3 // dim of matrix


//Fattened matrix multiplication . Kernel does not support x,y addressing
__global__ void mat_multiply(int* d_mat1, int* d_mat2, int* d_mat3, int width)
{
	int k,sum=0;
	int col = blockDim.x * blockIdx.x + threadIdx.x;
	int row = blockDim.y * blockIdx.y + threadIdx.y;

	if(row<width && col<width)
	{
		for(k=0;k<width;k++)
		{
			sum += d_mat1[row*width+k] * d_mat2[k*width+col];
		}
		d_mat3[row*width+col] = sum;
	}

}

int main()
{
	int i,j;
	int SIZE = N*N;
	//int BYTES = SIZE*sizeof(int);

	int *d_mat1, *d_mat2, *d_mat3;

	// allocate memory on the device
	hipMallocManaged(&d_mat1,N*N*sizeof(int));
	hipMallocManaged(&d_mat2,N*N*sizeof(int));
	hipMallocManaged(&d_mat3,N*N*sizeof(int));

	// generate matrix on host
	for(i=0;i<N*N;i++) //linearize array
	{
			d_mat1[i] = 1;
			d_mat2[i] = 1;
			d_mat3[i] = 0;

	}

	dim3 dimGrid(1,1);
	dim3 dimBlock(N,N);

	// lauch kernel
	mat_multiply<<<dimGrid,dimBlock>>>(d_mat1,d_mat2,d_mat3,N);
	hipDeviceSynchronize();

	for(i=0;i<N*N;i++)
	{
		
		printf("%d ",d_mat3[i]);
		if(i%N==0 && i>N)
			printf("\n");
	}
	printf("\n");
}